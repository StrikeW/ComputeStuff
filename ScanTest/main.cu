#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <cassert>

#include <Scan.h>

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

namespace {

  void logFailure(hipError_t error, const char *file, int line)
  {
    std::cerr << file << '@' << line << ": CUDA error: " << hipGetErrorName(error) << std::endl;
    abort();
  }
}
#define assertSuccess(a) do { hipError_t rv = (a); if(rv != hipSuccess) logFailure(rv, __FILE__, __LINE__); } while(0)


void runSize(uint32_t N)
{
  std::vector<uint32_t> offsets(N + 1);
  std::vector<uint32_t> counts(N);
  std::vector<uint32_t> offsetsGold(N + 1);
  std::vector<uint32_t> scratch;

  uint32_t* sum_h, * sum_d;
  assertSuccess(hipHostAlloc(&sum_h, sizeof(uint32_t), hipHostMallocMapped));
  assertSuccess(hipHostGetDevicePointer(&sum_d, sum_h, 0));


  uint32_t* offsets_d;
  uint32_t* scratch_d;
  uint32_t* counts_d;
  assertSuccess(hipMalloc(&offsets_d, sizeof(uint32_t)*(N + 1)));
  assertSuccess(hipMalloc(&scratch_d, ComputeStuff::Scan::scratchByteSize(N)));
  assertSuccess(hipMalloc(&counts_d, sizeof(uint32_t)*N));

  for (uint32_t modulo = 1; modulo < 10; modulo++) {
    std::cerr << "N=" << N << ", modulo=" << modulo << ", levels=" << ComputeStuff::Scan::levels(N) << ", scratch=" << ComputeStuff::Scan::scratchByteSize(N) / sizeof(uint32_t) << std::endl;

    offsetsGold[0] = 0;
    for (size_t i = 0; i < N; i++) {
      counts[i] = modulo == 1 ? 1 : (i % modulo);
      offsetsGold[i + 1] = offsetsGold[i] + counts[i];
    }
    assertSuccess(hipMemcpy(counts_d, counts.data(), sizeof(uint32_t)*N, hipMemcpyHostToDevice));

    ComputeStuff::Scan::calcOffsets(offsets_d, scratch_d, counts_d, N);
    assertSuccess(hipStreamSynchronize(0));
    assertSuccess(hipGetLastError());

#if 0
    scratch.resize(ComputeStuff::Scan::scratchByteSize(N) / sizeof(uint32_t));
    assertSuccess(hipMemcpy(scratch.data(), scratch_d, sizeof(uint32_t)*scratch.size(), hipMemcpyDeviceToHost));
#endif

    assertSuccess(hipMemcpy(offsets.data(), offsets_d, sizeof(uint32_t)*(N + 1), hipMemcpyDeviceToHost));
    for (size_t i = 0; i < N + 1; i++) {
      auto a = offsets[i];
      auto b = offsetsGold[i];
      assert(a == b);
    }

    ComputeStuff::Scan::calcOffsets(offsets_d, sum_d, scratch_d, counts_d, N);
    assertSuccess(hipStreamSynchronize(0));
    assertSuccess(hipGetLastError());

    assert(*((volatile uint32_t*)sum_h) == offsetsGold.back());

    assertSuccess(hipMemcpy(offsets.data(), offsets_d, sizeof(uint32_t)*(N + 1), hipMemcpyDeviceToHost));

    for (size_t i = 0; i < N + 1; i++) {
      assert(offsets[i] == offsetsGold[i]);
    }
  }

  assertSuccess(hipFree(counts_d));
  assertSuccess(hipFree(scratch_d));
  assertSuccess(hipFree(offsets_d));
  assertSuccess(hipHostFree(sum_h));
}


int main()
{
  assertSuccess(hipSetDevice(0));

  runSize(static_cast<uint32_t>(0u));
  for (uint64_t N = 0; N < (uint64_t)(1 << 31 - 1); N = (N == 0 ? 1 : 7 * N + N / 3))
  {
    runSize(static_cast<uint32_t>(N));
  }
  //runSize(1 << 31 - 1);


  return 0;
}
