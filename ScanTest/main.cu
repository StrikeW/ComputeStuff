#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <cassert>

#include <Scan.h>

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

namespace {

  void logFailure(hipError_t error, const char *file, int line)
  {
    std::cerr << file << '@' << line << ": CUDA error: " << hipGetErrorName(error) << std::endl;
    abort();
  }
}
#define assertSuccess(a) do { hipError_t rv = (a); if(rv != hipSuccess) logFailure(rv, __FILE__, __LINE__); } while(0)


int main()
{
  assertSuccess(hipSetDevice(0));

  // zero-copy mem for total sum readback.

  uint32_t* sum_h;
  assertSuccess(hipHostAlloc(&sum_h, sizeof(uint32_t), hipHostMallocMapped));

  uint32_t* sum_d;
  assertSuccess(hipHostGetDevicePointer(&sum_d, sum_h, 0));

  std::vector<uint32_t> offsets;
  std::vector<uint32_t> counts;
  std::vector<uint32_t> offsetsGold;

  std::vector<uint32_t> scratch;

  size_t N = 2 * 4 * 4 * 32;
  {
    uint32_t* offsets_d;
    uint32_t* scratch_d;
    uint32_t* counts_d;

    assertSuccess(hipMalloc(&offsets_d, sizeof(uint32_t)*(N + 1)));
    assertSuccess(hipMalloc(&scratch_d, ComputeStuff::Scan::scratchByteSize<uint32_t>(N)));
    assertSuccess(hipMalloc(&counts_d, sizeof(uint32_t)*N));

    counts.resize(N);
    offsetsGold.resize(N + 1);
    offsetsGold[0] = 0;
    offsets.resize(N + 1);

    for (uint32_t modulo = 1; modulo < 10; modulo++) {
      std::cerr << "N=" << N << ", modulo=" << modulo << " scratch=" << ComputeStuff::Scan::scratchByteSize<uint32_t>(N) / sizeof(uint32_t) <<  std::endl;
      for (size_t i = 0; i < N; i++) {
        counts[i] = modulo==1 ? 1 : (i % modulo);
        offsetsGold[i + 1] = offsetsGold[i] + counts[i];
      }
      assertSuccess(hipMemcpy(counts_d, counts.data(), sizeof(uint32_t)*N, hipMemcpyHostToDevice));

      ComputeStuff::Scan::calcOffsets(offsets_d, scratch_d, counts_d, N);
      assertSuccess(hipGetLastError());

#if 1
      scratch.resize(ComputeStuff::Scan::scratchByteSize<uint32_t>(N)/sizeof(uint32_t));
      assertSuccess(hipMemcpy(scratch.data(), scratch_d, sizeof(uint32_t)*scratch.size(), hipMemcpyDeviceToHost));
#endif

      assertSuccess(hipMemcpy(offsets.data(), offsets_d, sizeof(uint32_t)*(N + 1), hipMemcpyDeviceToHost));
      for (size_t i = 0; i < N + 1; i++) {
        assert(offsets[i] == offsetsGold[i]);
      }

      ComputeStuff::Scan::calcOffsets(offsets_d, sum_d, scratch_d, counts_d, N);
      assertSuccess(hipGetLastError());

      // Huh, hipStreamSynchronize for stream 0 is needed for sum_h to be in sync.
      // I thought stream 0 was in sync...
      assertSuccess(hipStreamSynchronize(0));
      assert(*((volatile uint32_t*)sum_h) == offsetsGold.back());

      assertSuccess(hipMemcpy(offsets.data(), offsets_d, sizeof(uint32_t)*(N + 1), hipMemcpyDeviceToHost));
      for (size_t i = 0; i < N + 1; i++) {
        assert(offsets[i] == offsetsGold[i]);
      }

    }

    assertSuccess(hipFree(counts_d));
    assertSuccess(hipFree(scratch_d));
    assertSuccess(hipFree(offsets_d));
  }

  assertSuccess(hipHostFree(sum_h));

  return 0;
}
