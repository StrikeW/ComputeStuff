#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/scan.h>
#include <thrust/execution_policy.h>
#include <iostream>
#include <vector>
#include <cassert>

#include <Scan.h>


#include <stdio.h>

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

namespace {

  void logFailure(hipError_t error, const char *file, int line)
  {
    std::cerr << file << '@' << line << ": CUDA error: " << hipGetErrorName(error) << std::endl;
    abort();
  }
}
#define assertSuccess(a) do { hipError_t rv = (a); if(rv != hipSuccess) logFailure(rv, __FILE__, __LINE__); } while(0)


void runTest(uint32_t N)
{
  std::vector<uint32_t> offsets(N + 1);
  std::vector<uint32_t> counts(N);
  std::vector<uint32_t> offsetsGold(N + 1);
  std::vector<uint32_t> scratch;

  uint32_t* sum_h, * sum_d;
  assertSuccess(hipHostAlloc(&sum_h, sizeof(uint32_t), hipHostMallocMapped));
  assertSuccess(hipHostGetDevicePointer(&sum_d, sum_h, 0));


  uint32_t* offsets_d;
  uint32_t* scratch_d;
  uint32_t* counts_d;
  assertSuccess(hipMalloc(&offsets_d, sizeof(uint32_t)*(N + 1)));
  assertSuccess(hipMalloc(&scratch_d, ComputeStuff::Scan::scratchByteSize(N)));
  assertSuccess(hipMalloc(&counts_d, sizeof(uint32_t)*N));

  for (uint32_t modulo = 1; modulo < 10; modulo++) {
    std::cerr << "N=" << N << ", modulo=" << modulo << ", levels=" << ComputeStuff::Scan::levels(N) << ", scratch=" << ComputeStuff::Scan::scratchByteSize(N) / sizeof(uint32_t) << std::endl;

    offsetsGold[0] = 0;
    for (size_t i = 0; i < N; i++) {
      counts[i] = modulo == 1 ? 1 : (i % modulo);
      offsetsGold[i + 1] = offsetsGold[i] + counts[i];
    }
    assertSuccess(hipMemcpy(counts_d, counts.data(), sizeof(uint32_t)*N, hipMemcpyHostToDevice));

    ComputeStuff::Scan::calcOffsets(offsets_d, scratch_d, counts_d, N);
    assertSuccess(hipStreamSynchronize(0));
    assertSuccess(hipGetLastError());

#if 0
    scratch.resize(ComputeStuff::Scan::scratchByteSize(N) / sizeof(uint32_t));
    assertSuccess(hipMemcpy(scratch.data(), scratch_d, sizeof(uint32_t)*scratch.size(), hipMemcpyDeviceToHost));
#endif

    assertSuccess(hipMemcpy(offsets.data(), offsets_d, sizeof(uint32_t)*(N + 1), hipMemcpyDeviceToHost));
    for (size_t i = 0; i < N + 1; i++) {
      auto a = offsets[i];
      auto b = offsetsGold[i];
      assert(a == b);
    }

    ComputeStuff::Scan::calcOffsets(offsets_d, sum_d, scratch_d, counts_d, N);
    assertSuccess(hipStreamSynchronize(0));
    assertSuccess(hipGetLastError());

    assert(*((volatile uint32_t*)sum_h) == offsetsGold.back());

    assertSuccess(hipMemcpy(offsets.data(), offsets_d, sizeof(uint32_t)*(N + 1), hipMemcpyDeviceToHost));

    for (size_t i = 0; i < N + 1; i++) {
      assert(offsets[i] == offsetsGold[i]);
    }
  }

  assertSuccess(hipFree(counts_d));
  assertSuccess(hipFree(scratch_d));
  assertSuccess(hipFree(offsets_d));
  assertSuccess(hipHostFree(sum_h));
}

void runPerf(uint32_t N)
{
  hipStream_t stream;
  assertSuccess(hipStreamCreate(&stream));

  hipEvent_t startA, stopA, startB, stopB;
  assertSuccess(hipEventCreate(&startA));
  assertSuccess(hipEventCreate(&startB));
  assertSuccess(hipEventCreate(&stopA));
  assertSuccess(hipEventCreate(&stopB));

  thrust::host_vector<uint32_t> in_h(N);
  std::vector<uint32_t> in_s(N);
  for (size_t i = 0; i < N; i++) {
    in_h[i] = in_s[i] = i % 3;
  }
  uint32_t* offsets_d;
  uint32_t* scratch_d;
  uint32_t* counts_d;
  assertSuccess(hipMalloc(&offsets_d, sizeof(uint32_t)*(N + 1)));
  assertSuccess(hipMalloc(&scratch_d, ComputeStuff::Scan::scratchByteSize(N)));
  assertSuccess(hipMalloc(&counts_d, sizeof(uint32_t)*N));
  assertSuccess(hipMemcpy(counts_d, in_s.data(), sizeof(uint32_t)*N, hipMemcpyHostToDevice));

  thrust::device_vector<uint32_t> in_d = in_h;
  thrust::device_vector<uint32_t> out_d(N);

  // Run thrust::exclusive_scan
  for (uint32_t i = 0; i < 10; i++) {  // warm-up
    thrust::exclusive_scan(thrust::cuda::par.on(stream), in_d.begin(), in_d.end(), out_d.begin());
  }
  hipEventRecord(startA, stream);
  for (uint32_t i = 0; i < 50; i++) {  // perf-run
    thrust::exclusive_scan(thrust::cuda::par.on(stream), in_d.begin(), in_d.end(), out_d.begin());
  }
  hipEventRecord(stopA, stream);

  // Run ComputeStuff scan
  for (uint32_t i = 0; i < 10; i++) {  // warm-up
    ComputeStuff::Scan::calcOffsets(offsets_d, scratch_d, counts_d, N);
  }
  hipEventRecord(startB, stream);
  for (uint32_t i = 0; i < 50; i++) {  // perf-run
    ComputeStuff::Scan::calcOffsets(offsets_d, scratch_d, counts_d, N);
  }
  hipEventRecord(stopB, stream);

  hipEventSynchronize(stopB);
  float elapsedA, elapsedB;
  assertSuccess(hipEventElapsedTime(&elapsedA, startA, stopA));
  assertSuccess(hipEventElapsedTime(&elapsedB, startB, stopB));

  std::cerr << "N=" << N << ",\tthrust=" << (elapsedA / 50.0) << "ms,\tComputeStuff=" << (elapsedB / 50.0) << "ms,\tratio CS/thrust=" << (elapsedB/elapsedA) << std::endl;
 

  assertSuccess(hipFree(counts_d));
  assertSuccess(hipFree(scratch_d));
  assertSuccess(hipFree(offsets_d));



  assertSuccess(hipStreamDestroy(stream));
  assertSuccess(hipEventDestroy(startA));
  assertSuccess(hipEventDestroy(startB));
  assertSuccess(hipEventDestroy(stopA));
  assertSuccess(hipEventDestroy(stopB));
}

int main(int argc, char** argv)
{
  bool perf = true;
  bool test = false;
  for (int i = 1; i < argc; i++) {
    if (strcmp("--perf", argv[i])) {
      perf = true;
    }
    else if (strcmp("--no-perf", argv[i])) {
      perf = false;
    }
    else if (strcmp("--test", argv[i])) {
      test = true;
    }
    else if (strcmp("--no-test", argv[i])) {
      test = false;
    }
  }

  assertSuccess(hipSetDevice(0));

  if (test) {
    runTest(static_cast<uint32_t>(0u));
    for (uint64_t N = 1; N < (uint64_t)(1 << 31 - 1); N = (N == 0 ? 1 : 7 * N + N / 3))
    {
      runTest(static_cast<uint32_t>(N));
    }
    //runSize(1 << 31 - 1);
  }

  if (perf) {
    for (uint64_t N = 1; N < (uint64_t)(1 << 29 - 1); N = 3 * N + N / 3) {
      runPerf(static_cast<uint32_t>(N));
    }
  }


  return 0;
}
