#include "hip/hip_runtime.h"
#include <array>
#include <cstdint>
#include <vector>
#include <hip/hip_runtime.h>
#include <cassert>
#include <>

#include "HP5.h"

#define HP5_WARP_COUNT 5
#define HP5_WARP_SIZE 32

namespace {


  __global__ __launch_bounds__(128) void reduceBase(uint32_t* __restrict__ hp_d,
                                                    uint32_t* __restrict__ sb_d,
                                                    const uint32_t n1,
                                                    const uint32_t* __restrict__ src,
                                                    const uint32_t n0)
  {
    const uint32_t offset0 = blockDim.x * blockIdx.x + threadIdx.x;
    const uint32_t lane = threadIdx.x %  HP5_WARP_SIZE;
    const uint32_t value = offset0 < n0 ? src[offset0] : 0;
    const uint32_t warpMask = __ballot(value != 0);
    if (lane == 0) {
      const uint32_t offset1 = offset0 / HP5_WARP_SIZE;
      if (offset1 < n1) {
        hp_d[offset1] = warpMask;
        sb_d[offset1] = __popc(warpMask);
      }
    }
  }

  __global__ __launch_bounds__(256) void reduceBase2(uint32_t* __restrict__ hp2_d,
                                                     uint32_t* __restrict__ sb2_d,
                                                     const uint32_t n2,
                                                     uint32_t* __restrict__ hp1_d,
                                                     const uint32_t n1,
                                                     const uint32_t* __restrict__ sb0_d,
                                                     const uint32_t n0)
  {
    const uint32_t offset0_base = 32 * 160 * blockIdx.x + threadIdx.x;
    const uint32_t offset1_base = 160 * blockIdx.x;

    const uint32_t lane = threadIdx.x %  HP5_WARP_SIZE;
    const uint32_t warp = threadIdx.x / HP5_WARP_SIZE;

    __shared__ uint32_t sb1[160];

    for (uint32_t i = 0; i < 20; i++) {
      const uint32_t offset0 = offset0_base + 256 * i;
      const uint32_t value = offset0 < n0 ? sb0_d[offset0] : 0;
      const uint32_t warpMask = __ballot(value != 0);
      if (lane == 0) {
        const uint32_t offset1 = offset1_base + 8 * i + warp;
        if (offset1 < n1) {
          hp1_d[offset1] = warpMask;
        }
        sb1[8 * i + warp] = __popc(warpMask);
      }
    }

    __syncthreads();
    if (threadIdx.x < HP5_WARP_SIZE) { // First warp
      const uint32_t offset2 = 32 * blockIdx.x + threadIdx.x;
      if (offset2 < n2) {
        uint4 hp = make_uint4(sb1[5 * threadIdx.x + 0],
                              sb1[5 * threadIdx.x + 1],
                              sb1[5 * threadIdx.x + 2],
                              sb1[5 * threadIdx.x + 3]);
        ((uint4*)hp2_d)[offset2] = hp;
        sb2_d[offset2] = hp.x + hp.y + hp.z + hp.w + sb1[5 * threadIdx.x + 4];
      }
    }

  }

  // Reads 160 values, outputs HP level of 128 values, and 32 sideband values.
  __global__
  __launch_bounds__(HP5_WARP_COUNT * HP5_WARP_SIZE)
  void
  reduce1(uint4* __restrict__ hp1_d,
          uint32_t* __restrict__ sb1_d,
          const uint32_t n1,
          const uint32_t* __restrict__ sb0_d,
          const uint32_t n0)
  {
    const uint32_t offset0 = HP5_WARP_COUNT * HP5_WARP_SIZE * blockIdx.x + threadIdx.x;

    // Idea, each warp reads 32 values. read instead 32/4 uint4's.

    __shared__ uint32_t sb[HP5_WARP_COUNT * HP5_WARP_SIZE];
    sb[threadIdx.x] = offset0 < n0 ? sb0_d[offset0] : 0;

    __syncthreads();
    if (threadIdx.x < HP5_WARP_SIZE) { // First warp
      const uint32_t offset1 = 32 * blockIdx.x + threadIdx.x;
      if (offset1 < n1) {
        uint4 hp = make_uint4(sb[5 * threadIdx.x + 0],
                              sb[5 * threadIdx.x + 1],
                              sb[5 * threadIdx.x + 2],
                              sb[5 * threadIdx.x + 3]);
        hp1_d[offset1] = hp;
        sb1_d[offset1] = hp.x + hp.y + hp.z + hp.w + sb[5 * threadIdx.x + 4];
      }
    }
  }

  template<bool mask_input, bool write_sum>
  __global__
  __launch_bounds__(128)
  void
  reduceApex(uint4* __restrict__ apex_d,
             uint32_t* sum_d,
             const uint32_t* in_d,
             uint32_t N,
             uint32_t offset0,
             uint32_t offset1,
             uint32_t offset2,
             uint32_t offset3,
             uint32_t offset4,
             uint32_t offset5,
             uint32_t offset6,
             uint32_t offset7,
             uint32_t offset8,
             uint32_t offset9,
             uint32_t offsetA,
             uint32_t offsetB,
             uint32_t offsetC)
  {
    // 0 : sum + 3 padding
    // 1 : 1 uvec4 of level 0.
    // 2 : 5 values of level 0 (top)
    // 7 : 25 values of level 1
    // 32: total sum.

    // Fetch up to 125 elements from in_d.
    uint32_t a = threadIdx.x < N ? in_d[threadIdx.x] : 0;
    if (mask_input) {
      a = a != 0 ? 1 : 0;
    }
    volatile __shared__ uint32_t sb[125 + 25];
    sb[threadIdx.x] = a;

    // Store 5x5 uint4's at uint4 offset 0 (25x4=100 elements, corresponding to 125 inputs).
    __syncthreads();

    if (threadIdx.x < 32) {

      if (threadIdx.x < 25) {
        uint32_t e0 = sb[5 * threadIdx.x + 0];
        uint32_t e1 = sb[5 * threadIdx.x + 1];
        uint32_t e2 = sb[5 * threadIdx.x + 2];
        uint32_t e3 = sb[5 * threadIdx.x + 3];
        uint32_t e4 = sb[5 * threadIdx.x + 4];
        apex_d[7 + threadIdx.x] = make_uint4(e0,
                                             e0 + e1,
                                             e0 + e1 + e2,
                                             e0 + e1 + e2 + e3);

        sb[125 + threadIdx.x] = e0 + e1 + e2 + e3 + e4;
      }

      // Store 5 uint4's at uint4 offset 25 (5x4=20 elements, corresponding to 25 inputs).
      //__syncthreads();
      if (threadIdx.x < 5) {
        uint32_t e0 = sb[125 + 5 * threadIdx.x + 0];
        uint32_t e1 = sb[125 + 5 * threadIdx.x + 1];
        uint32_t e2 = sb[125 + 5 * threadIdx.x + 2];
        uint32_t e3 = sb[125 + 5 * threadIdx.x + 3];
        uint32_t e4 = sb[125 + 5 * threadIdx.x + 4];
        apex_d[2 + threadIdx.x] = make_uint4(e0,
                                             e0 + e1,
                                             e0 + e1 + e2,
                                             e0 + e1 + e2 + e3);

        sb[threadIdx.x] = e0 + e1 + e2 + e3 + e4;
      }

      // Store 1 uint4 at uint4 offset 30 (1x4=4 elements, corresponding to 5 inputs)
      // Store total at uint4 offset 31
      //__syncthreads();
      if (threadIdx.x < 1) {
        uint32_t e0 = sb[0];
        uint32_t e1 = sb[1];
        uint32_t e2 = sb[2];
        uint32_t e3 = sb[3];
        uint32_t e4 = sb[4];
        apex_d[1 + threadIdx.x] = make_uint4(e0,
                                             e0 + e1,
                                             e0 + e1 + e2,
                                             e0 + e1 + e2 + e3);
        uint32_t s = e0 + e1 + e2 + e3 + e4;
        *reinterpret_cast<uint32_t*>(apex_d) = s;
        if (write_sum) {
          *sum_d = s;
        }
      }
    }
    else if (threadIdx.x == 32) {
      uint32_t * t = (uint32_t*)(apex_d + 32);

      if (offset0 != 0) t[0] = offset0;
      if (offset1 != 0) t[1] = offset1;
      if (offset2 != 0) t[2] = offset2;
      if (offset3 != 0) t[3] = offset3;
      if (offset4 != 0) t[4] = offset4;
      if (offset5 != 0) t[5] = offset5;
      if (offset6 != 0) t[6] = offset6;
      if (offset7 != 0) t[7] = offset7;
      if (offset8 != 0) t[8] = offset8;
      if (offset9 != 0) t[9] = offset9;
      if (offsetA != 0) t[10] = offsetA;
      if (offsetB != 0) t[11] = offsetB;
      if (offsetC != 0) t[12] = offsetC;
    }
  }

  /*template<uint32_t L>
  struct LevelOffsets
  {
    __host__ LevelOffsets(const LevelOffsets&) = default;
    __host__ LevelOffsets(uint32_t* v) { for (uint32_t i = 0; i < L; i++) value[i] = v[i]; }
    uint32_t value[L];
  };*/

  __device__ __forceinline__ uint32_t processHistoElement(uint32_t& key, uint32_t offset, const uint4 element)
  {
    assert(element.x <= element.y);
    assert(element.y <= element.z);
    assert(element.z <= element.w);

    if (key < element.x) {
    }
    else if (key < element.y) {
      key -= element.x;
      offset += 1;
    }
    else if (key < element.z) {
      key -= element.y;
      offset += 2;
    }
    else if (key < element.w) {
      key -= element.z;
      offset += 3;
    }
    else  {
      key -= element.w;
      offset += 4;
    }
    return offset;
  }

  __device__ __forceinline__ uint32_t processDataElement(uint32_t& key, uint32_t offset, const uint4 element)
  {
    if (element.x <= key) {
      key -= element.x;
      offset++;
      if (element.y <= key) {
        key -= element.y;
        offset++;
        if (element.z <= key) {
          key -= element.z;
          offset++;
          if (element.w <= key) {
            key -= element.w;
            offset++;
          }
        }
      }
    }
    return offset;
  }

  __device__ __forceinline__ uint32_t processMaskElement(uint32_t& key, uint32_t offset, uint32_t mask)
  {
    const uint32_t m16 = 0xffffu;
    const uint32_t c16 = __popc(mask & m16);
    if (c16 <= key) {  // Key is in upper 16 bits
      key -= c16;
      offset += 16;
      mask = mask >> 16;
    }
    const uint32_t m8 = 0xffu;
    const uint32_t c8 = __popc(mask & m8);
    if(c8 <= key) { // Key is in upper 8 bits
      key -= c8;
      offset += 8;
      mask = mask >> 8;
    }
    const uint32_t m4 = 0xfu;
    const uint32_t c4 = __popc(mask & m4);
    if (c4 <= key) { // Key is in upper 4 bits
      key -= c4;
      offset += 4;
      mask = mask >> 4;
    }
    const uint32_t m2 = 0x3u;
    const uint32_t c2 = __popc(mask & m2);
    if (c2 <= key) { // Key is in upper 2 bits
      key -= c2;
      offset += 2;
      mask = mask >> 2;
    }

    if ((mask & 0x1) <= key) {
      offset++;
    }
    return offset;
  }

  struct OffsetBlob
  {
    uint32_t data0;
    uint32_t data1;
    uint32_t data2;
    uint32_t data3;
    uint32_t data4;
    uint32_t data5;
    uint32_t data6;
    uint32_t data7;
    uint32_t data8;
    uint32_t data9;
    uint32_t dataA;
    uint32_t dataB;
  };

  __device__ __forceinline__ uint32_t _ldu(const uint32_t *ptr)
  {
    uint32_t rv;
    asm ("ldu.global.u32 %0, [%1];" : "=r"(rv) :
#if defined(__LP64__) || defined(_WIN64)
                  "l"(ptr)
#else
                  "r"(ptr)
#endif
    );
    return rv;
  }

  __device__ __forceinline__ uint4 _ldu(const uint4 *ptr)
  {
    uint4 rv;
    asm("ldu.global.v4.u32 {%0,%1,%2,%3}, [%4];" : "=r"(rv.x), "=r"(rv.y), "=r"(rv.z), "=r"(rv.w) :
#if defined(__LP64__) || defined(_WIN64)
        "l"(ptr)
#else
        "r"(ptr)
#endif
    );
    return rv;
  }


  template<uint32_t L>
  __global__
  __launch_bounds__(128)
  void extract(uint32_t* __restrict__ out_d,
               const uint32_t* __restrict__ hp_d)
  {
    uint32_t N = hp_d[0];
    uint4 T = *((const uint4*)(hp_d + 4));

    for (uint32_t k = 4 * blockDim.x * blockIdx.x; k < N; k += 4 * gridDim.x * blockDim.x) {
      uint32_t index = k + 4 * threadIdx.x;
      if (index < N) {
        uint4 offset = make_uint4(0, 0, 0, 0);
        uint4 key = make_uint4(index, index + 1, index + 2, index + 3);
        if (N <= key.w) {
          key.y = min(key.y, N - 1);
          key.z = min(key.z, N - 1);
          key.w = min(key.w, N - 1);
        }

        offset.x = processHistoElement(key.x, 5 * offset.x, T);
        offset.y = processHistoElement(key.y, 5 * offset.y, T);
        offset.z = processHistoElement(key.z, 5 * offset.z, T);
        offset.w = processHistoElement(key.w, 5 * offset.w, T);

        offset.x = processHistoElement(key.x, 5 * offset.x, *(const uint4*)(hp_d + 8 + 4 * offset.x));
        offset.y = processHistoElement(key.y, 5 * offset.y, *(const uint4*)(hp_d + 8 + 4 * offset.y));
        offset.z = processHistoElement(key.z, 5 * offset.z, *(const uint4*)(hp_d + 8 + 4 * offset.z));
        offset.w = processHistoElement(key.w, 5 * offset.w, *(const uint4*)(hp_d + 8 + 4 * offset.w));

        offset.x = processHistoElement(key.x, 5 * offset.x, *(const uint4*)(hp_d + 28 + 4 * offset.x));
        offset.y = processHistoElement(key.y, 5 * offset.y, *(const uint4*)(hp_d + 28 + 4 * offset.y));
        offset.z = processHistoElement(key.z, 5 * offset.z, *(const uint4*)(hp_d + 28 + 4 * offset.z));
        offset.w = processHistoElement(key.w, 5 * offset.w, *(const uint4*)(hp_d + 28 + 4 * offset.w));

        for (uint32_t i = L; 1 < i; i--) {
          uint32_t offseti = *(hp_d + 32 * 4 + i - 1);
          offset.x = processDataElement(key.x, 5 * offset.x, *(const uint4*)(hp_d + offseti + 4 * offset.x));
          offset.y = processDataElement(key.y, 5 * offset.y, *(const uint4*)(hp_d + offseti + 4 * offset.y));
          offset.z = processDataElement(key.z, 5 * offset.z, *(const uint4*)(hp_d + offseti + 4 * offset.z));
          offset.w = processDataElement(key.w, 5 * offset.w, *(const uint4*)(hp_d + offseti + 4 * offset.w));
        }
        uint32_t offset0 = *(hp_d + 32 * 4);
        uint4 val;
        val.x = processMaskElement(key.x, 32 * offset.x, hp_d[offset0 + offset.x]);
        val.y = processMaskElement(key.y, 32 * offset.y, hp_d[offset0 + offset.y]);
        val.z = processMaskElement(key.z, 32 * offset.z, hp_d[offset0 + offset.z]);
        val.w = processMaskElement(key.w, 32 * offset.w, hp_d[offset0 + offset.w]);

        if (index + 3 < N) {
          *(uint4*)(out_d + index) = val;
        }
        else {
          out_d[index] = val.x;
          if (index + 1 < N) {
            out_d[index + 1] = val.y;
            if (index + 2 < N) {
              out_d[index + 2] = val.z;
            }
          }
        }
        //uint32_t * dst = out_d + index;
//#if 0
//        *dst = val;
//#else
//        asm("st.global.cs.u32 [%0], %1;" ::  "l"(dst), "r"(val));
//#endif

      }
    }
  }


  template<uint32_t L>
  void runExtract(uint32_t* out_d, const std::vector<uint32_t>& offsets, const uint32_t* hp_d, uint32_t N, hipStream_t stream)
  {
    int minGridSize = 0;
    int blockSize = 0;
    auto rv = hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, extract<L>);
    //assert(rv == hipSuccess);

    auto blocks = std::min(minGridSize, int((N + 4 * blockSize - 1) / (4 * blockSize)));
    ::extract<L><<<blocks, blockSize, 0, stream>>>(out_d, hp_d);
  }

  void scratchLayout(std::vector<uint32_t>& levels, std::vector<uint32_t>& offsets, uint32_t N)
  {
    if (N == 0) return;
    
    levels.clear();
    levels.push_back((N + 31) / 32);
    while (125 < levels.back())
    {
      levels.push_back((levels.back() + 4) / 5);
    }
    
    offsets.resize(levels.size() + 4);

    uint32_t o = 0;
    offsets[levels.size()] = o;                 // Apex
    o += 32 * 4;                                // Size of apex
    o += static_cast<uint32_t>(levels.size());  // Space for level offsets
    o = (o + 3) & ~3;                           // Align to uvec4

    for (int i = static_cast<int>(levels.size()) - 1; 0 < i; i--) {
      offsets[i] = o; // HP level i
      o += 4 * levels[i];
    }

    // level zero
    offsets[0] = o;
    o += (levels[0] + 3) & ~3;

    offsets[levels.size() + 1] = o; // Large sideband buffer
    o += levels.empty() ? 0 : (levels[0] + 3) & ~3;

    offsets[levels.size() + 2] = o; // Small sideband buffer
    o += levels.size() < 2 ? 0 : (levels[1] + 3) & ~3;

    offsets[levels.size() + 3] = o; // Final size
  }

  


}

size_t ComputeStuff::HP5::scratchByteSize(uint32_t N)
{
  std::vector<uint32_t> levels;
  std::vector<uint32_t> offsets;
  scratchLayout(levels, offsets, N);
  return 4 * sizeof(uint32_t)*offsets.back();
}

void ComputeStuff::HP5::compact(uint32_t* out_d,
                                uint32_t* sum_d,
                                uint32_t* scratch_d,
                                const uint32_t* in_d,
                                uint32_t N,
                                hipStream_t stream)
{
  if (N == 0) return;

  std::vector<uint32_t> levels;
  std::vector<uint32_t> offsets;
  scratchLayout(levels, offsets, N);

  auto L = levels.size();
  if (L == 0) {
    assert(false);
  }
  else {
    bool sb = false;

    size_t i = 0;
    if (1 < L) {
      ::reduceBase2<<<(levels[1] + 31) / 32, 8 * 32, 0, stream>>>(scratch_d + offsets[1],
                                                                  scratch_d + offsets[L + 1 + (sb ? 1 : 0)],
                                                                  levels[1],
                                                                  scratch_d + offsets[0],
                                                                  levels[0],
                                                                  in_d,
                                                                  N);
      i += 2;
      sb = !sb;
    }
    else if (0 < L) {
      ::reduceBase << <(levels[0] + 3) / 4, 4 * 32, 0, stream >> > (scratch_d + offsets[0],
                                                                    scratch_d + offsets[L + 1 + (sb ? 1 : 0)],
                                                                    levels[0],
                                                                    in_d,
                                                                    N);
      i += 1;
      sb = !sb;
    }


    for (; i < L; i++) {
      ::reduce1<<<(levels[i] + 31)/32, 160, 0, stream>>>(reinterpret_cast<uint4*>(scratch_d + offsets[i]),
                                                         scratch_d + offsets[L + 1 + (sb ? 1 : 0)],
                                                         levels[i],
                                                         scratch_d + offsets[L + 1 + (sb ? 0 : 1)],
                                                         levels[i - 1]);
      sb = !sb;
    }
    ::reduceApex<false, true><<<1, 128, 0, stream>>>(reinterpret_cast<uint4*>(scratch_d),
                                                     sum_d,
                                                     scratch_d + offsets[L + 1 + (sb ? 0 : 1)],
                                                     levels[L - 1],
                                                     0 < L ? offsets[0] : 0,
                                                     1 < L ? offsets[1] : 0,
                                                     2 < L ? offsets[2] : 0,
                                                     3 < L ? offsets[3] : 0,
                                                     4 < L ? offsets[4] : 0,
                                                     5 < L ? offsets[5] : 0,
                                                     6 < L ? offsets[6] : 0,
                                                     7 < L ? offsets[7] : 0,
                                                     8 < L ? offsets[8] : 0,
                                                     9 < L ? offsets[9] : 0,
                                                     10 < L ? offsets[10] : 0,
                                                     11 < L ? offsets[11] : 0,
                                                     12 < L ? offsets[12] : 0);
  }



 

  // No readback, no dynamic parallelism approach: Create enough blocks s.t. all multiprocessors have enough warps,
  // but let problem size beyond this be handled by looping. 

  switch (L)
  {
  case 0: ::runExtract<0>(out_d, offsets, scratch_d, N, stream); break;
  case 1: ::runExtract<1>(out_d, offsets, scratch_d, N, stream); break;
  case 2: ::runExtract<2>(out_d, offsets, scratch_d, N, stream); break;
  case 3: ::runExtract<3>(out_d, offsets, scratch_d, N, stream); break;
  case 4: ::runExtract<4>(out_d, offsets, scratch_d, N, stream); break;
  case 5: ::runExtract<5>(out_d, offsets, scratch_d, N, stream); break;
  case 6: ::runExtract<6>(out_d, offsets, scratch_d, N, stream); break;
  case 7: ::runExtract<7>(out_d, offsets, scratch_d, N, stream); break;
  case 8: ::runExtract<8>(out_d, offsets, scratch_d, N, stream); break;
  case 9: ::runExtract<9>(out_d, offsets, scratch_d, N, stream); break;
  case 10: ::runExtract<10>(out_d, offsets, scratch_d, N, stream); break;
  case 11: ::runExtract<11>(out_d, offsets, scratch_d, N, stream); break;
  case 12: ::runExtract<12>(out_d, offsets, scratch_d, N, stream); break;
  default:
    abort();
    break;
  }

#if 0
  hipStreamSynchronize(stream);
  auto error = hipGetLastError();
  if (error != hipSuccess) {
    abort();
  }
#endif
}
