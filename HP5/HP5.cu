#include "hip/hip_runtime.h"
#include <vector>
#include <hip/hip_runtime.h>
#include <>

#include "HP5.h"

#define HP5_WARP_COUNT 5
#define HP5_WARP_SIZE 32

namespace {

  // Reads 160 values, outputs HP level of 128 values, and 32 sideband values.

  template<bool mask_input>
  __global__
  __launch_bounds__(HP5_WARP_COUNT * HP5_WARP_SIZE)
  void
  reduce1(uint4* __restrict__ hp1_d,
          uint32_t* __restrict__ sb1_d,
          const uint32_t* __restrict__ sb0_d,
          uint32_t N)
  {
    const uint32_t threadOffset = HP5_WARP_COUNT * HP5_WARP_SIZE * blockIdx.x + threadIdx.x;

    // Idea, each warp reads 32 values. read instead 32/4 uint4's.

    __shared__ uint32_t sb[HP5_WARP_COUNT * HP5_WARP_SIZE];
    uint32_t a = threadOffset < N ? sb0_d[threadOffset] : 0;
    if (mask_input) {
      a = a != 0 ? 1 : 0;
    }
    sb[threadIdx.x] = a;

    __syncthreads();
    if (threadIdx.x < HP5_WARP_SIZE) { // First warp
      uint4 hp = make_uint4(sb[5 * threadIdx.x + 0],
                            sb[5 * threadIdx.x + 1],
                            sb[5 * threadIdx.x + 2],
                            sb[5 * threadIdx.x + 3]);
      hp1_d[32 * blockIdx.x + threadIdx.x] = hp;
      sb1_d[32 * blockIdx.x + threadIdx.x] = hp.x + hp.y + hp.z + hp.w + sb[5 * threadIdx.x + 4];
    }
  }

  template<bool mask_input, bool write_sum>
  __global__
  __launch_bounds__(128)
  void
  reduceApex(uint4* __restrict__ apex_d,
             uint32_t* sum_d,
             const uint32_t* in_d,
             uint32_t N)
  {
    // 0 : sum + 3 padding
    // 1 : 1 uvec4 of level 0.
    // 2 : 5 values of level 0 (top)
    // 7 : 25 values of level 1
    // 32: total sum.

    // Fetch up to 125 elements from in_d.
    uint32_t a = threadIdx.x < N ? in_d[threadIdx.x] : 0;
    if (mask_input) {
      a = a != 0 ? 1 : 0;
    }
    __shared__ uint32_t sb[125 + 25];
    sb[threadIdx.x] = a;

    // Store 5x5 uint4's at uint4 offset 0 (25x4=100 elements, corresponding to 125 inputs).
    __syncthreads();
    if (threadIdx.x < 25) {
      uint32_t e0 = sb[5 * threadIdx.x + 0];
      uint32_t e1 = sb[5 * threadIdx.x + 1];
      uint32_t e2 = sb[5 * threadIdx.x + 2];
      uint32_t e3 = sb[5 * threadIdx.x + 3];
      uint32_t e4 = sb[5 * threadIdx.x + 4];
      apex_d[7 + threadIdx.x] = make_uint4(e0,
                                       e0 + e1,
                                       e0 + e1 + e2,
                                       e0 + e1 + e2 + e3);

      sb[125 + threadIdx.x] = e0 + e1 + e2 + e3 + e4;
    }

    // Store 5 uint4's at uint4 offset 25 (5x4=20 elements, corresponding to 25 inputs).
    __syncthreads();
    if (threadIdx.x < 5) {
      uint32_t e0 = sb[125 + 5 * threadIdx.x + 0];
      uint32_t e1 = sb[125 + 5 * threadIdx.x + 1];
      uint32_t e2 = sb[125 + 5 * threadIdx.x + 2];
      uint32_t e3 = sb[125 + 5 * threadIdx.x + 3];
      uint32_t e4 = sb[125 + 5 * threadIdx.x + 4];
      apex_d[2 + threadIdx.x] = make_uint4(e0,
                                            e0 + e1,
                                            e0 + e1 + e2,
                                            e0 + e1 + e2 + e3);

      sb[threadIdx.x] = e0 + e1 + e2 + e3 + e4;
    }

    // Store 1 uint4 at uint4 offset 30 (1x4=4 elements, corresponding to 5 inputs)
    // Store total at uint4 offset 31
    __syncthreads();
    if (threadIdx.x < 1) {
      uint32_t e0 = sb[0];
      uint32_t e1 = sb[1];
      uint32_t e2 = sb[2];
      uint32_t e3 = sb[3];
      uint32_t e4 = sb[4];
      apex_d[1 + threadIdx.x] = make_uint4(e0,
                                            e0 + e1,
                                            e0 + e1 + e2,
                                            e0 + e1 + e2 + e3);
      uint32_t s = e0 + e1 + e2 + e3 + e4;
      *reinterpret_cast<uint32_t*>(apex_d) = s;
      if (write_sum) {
        *sum_d = s;
      }
    }

  }

  void scratchLayout(std::vector<uint32_t>& levels, std::vector<uint32_t>& offsets, uint32_t N)
  {
    if (N == 0) return;

    // Apex-level is always present.
    // Levels below apex, reduction is done in 160 -> 32 blocks.
    while (125 < N) {
      levels.push_back((N + 159) / 160);  // Number of blocks per level
      N = 32 * levels.back();
    }


    offsets.resize(levels.size() + 4);

    uint32_t o = 0;
    offsets[levels.size()] = o;  // Apex
    o += 128;

    for (int i = static_cast<int>(levels.size()) - 1; 0 <= i; i--) {
      offsets[i] = o; // HP level i
      o += 32 * 4 * levels[i];
    }
    offsets[levels.size() + 1] = o; // Large sideband buffer
    o += 32 * (levels.empty() ? 0 : levels[0]);

    offsets[levels.size() + 2] = o; // Small sideband buffer
    o += 32 * (levels.size() < 2 ? 0 : levels[1]);

    offsets[levels.size() + 3] = o; // Final size
  }

}

size_t ComputeStuff::HP5::scratchByteSize(uint32_t N)
{
  std::vector<uint32_t> levels;
  std::vector<uint32_t> offsets;
  scratchLayout(levels, offsets, N);
  return sizeof(uint32_t)*offsets.back();
}

void ComputeStuff::HP5::compact(uint32_t* out_d,
                                uint32_t* sum_d,
                                uint32_t* scratch_d,
                                const uint32_t* in_d,
                                uint32_t N,
                                hipStream_t stream)
{
  if (N == 0) return;

  std::vector<uint32_t> levels;
  std::vector<uint32_t> offsets;
  scratchLayout(levels, offsets, N);

  if (levels.empty()) {
    reduceApex<true, true> << <1, 128, 0, stream >> > (reinterpret_cast<uint4*>(scratch_d), sum_d, in_d, N);
  }
  else {
    abort();
  }
  hipStreamSynchronize(stream);
  auto error = hipGetLastError();
  if (error != hipSuccess) {
    abort();
  }


/*

  auto L = static_cast<uint32_t>(levels.size());



  ::reduce1<true><<<(N+159)/160, 160, 0, stream>>>(reinterpret_cast<uint4*>(scratch_d + offsets[0]),
                                                   scratch_d + offsets[L + 1],
                                                   in_d,
                                                   N);
  for (uint32_t l = 1; l < L; l++) {
    ::reduce1<false><<<(levels[l - 1] + 159) / 160, 160, 0, stream>>>(reinterpret_cast<uint4*>(scratch_d + offsets[L + 1 + ((l + 1) & 1)]),
                                                                      scratch_d + offsets[L + 1 + (l & 1)],
                                                                      in_d,
                                                                      levels[l - 1]);
  }

  */
}
