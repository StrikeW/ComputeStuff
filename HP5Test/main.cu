// This file is part of ComputeStuff copyright (C) 2017 Christopher Dyken.
// Released under the MIT license, please see LICENSE file for details.

#include <vector>
#include <iomanip>
#include <iostream>
#include <hip/hip_runtime.h>
#include <>

#include <HP5.h>
#include <Scan.h>

namespace {

  void logFailure(hipError_t error, const char *file, int line)
  {
    std::cerr << file << '@' << line << ": CUDA error: " << hipGetErrorName(error) << std::endl;
    abort();
  }
}
#define assertSuccess(a) do { hipError_t rv = (a); if(rv != hipSuccess) logFailure(rv, __FILE__, __LINE__); } while(0)

void assertMatching(const uint32_t* result, const uint32_t* gold, uint32_t N)
{
  for (size_t i = 0; i < N; i++) {
    auto a = result[i];
    auto b = gold[i];
    if (a != b) {
      std::cerr << "a=" << a << " !=  b=" << b << std::endl;
      abort();
    }
  }
}

void assertMatching(volatile uint32_t* a, uint32_t b)
{
  if (*a != b) {
    std::cerr << "a=" << *a << ", b=" << b << std::endl;
    abort();
  }
}

void buildCompactProblemWorstCase(std::vector<uint32_t>& out, uint32_t& sum, std::vector<uint32_t>& in, uint32_t N, uint32_t m)
{
  sum = 0;
  in.resize(N);
  out.resize(N);
  for (uint32_t i = 0; i < N; i++) {
    in[i] = m == 1 ? 1 : (i % m);
    if (in[i]) {
      out[sum++] = i;
    }
  }
}

void buildCompactProblemBestCase(std::vector<uint32_t>& out, uint32_t& sum, std::vector<uint32_t>& in, uint32_t N, uint32_t m)
{
  sum = 0;
  in.resize(N);
  out.resize(N);

  auto s = (N + m - 1) / m;
  for (uint32_t i = 0; i < s; i++) {
    in[i] = 1;
    out[sum++] = i;
  }
  for (uint32_t i = s; i < N; i++) {
    in[i] = 0;
  }
}

void runCompactTest(uint32_t N, uint32_t m)
{
  hipStream_t stream;
  assertSuccess(hipStreamCreate(&stream));

  hipEvent_t startA, stopA, startB, stopB, startC, stopC, startD, stopD;
  assertSuccess(hipEventCreate(&startA));
  assertSuccess(hipEventCreate(&startB));
  assertSuccess(hipEventCreate(&startC));
  assertSuccess(hipEventCreate(&startD));
  assertSuccess(hipEventCreate(&stopA));
  assertSuccess(hipEventCreate(&stopB));
  assertSuccess(hipEventCreate(&stopC));
  assertSuccess(hipEventCreate(&stopD));

  uint32_t* sum_h, *sum_d;
  assertSuccess(hipHostAlloc(&sum_h, sizeof(uint32_t), hipHostMallocMapped));
  assertSuccess(hipHostGetDevicePointer(&sum_d, sum_h, 0));

  uint32_t *out_d, *in_d, *hp5_scratch_d, *scan_scratch_d;
  assertSuccess(hipMalloc(&out_d, sizeof(uint32_t)*N));
  assertSuccess(hipMalloc(&in_d, sizeof(uint32_t)*N));
  assertSuccess(hipMalloc(&hp5_scratch_d, ComputeStuff::HP5::scratchByteSize(N)));
  assertSuccess(hipMalloc(&scan_scratch_d, ComputeStuff::Scan::scratchByteSize(N)));

  std::vector<uint32_t> out_h(N);

  uint32_t sum;
  std::vector<uint32_t> out, in;

  // Best case
  buildCompactProblemBestCase(out, sum, in, N, m);
  assertSuccess(hipMemcpy(in_d, in.data(), sizeof(uint32_t)*N, hipMemcpyHostToDevice));
  *sum_h = ~0u;
  for (uint32_t i = 0; i < 10; i++) {
    ComputeStuff::Scan::compact(out_d, sum_d, scan_scratch_d, in_d, N, stream);
  }
  hipEventRecord(startA, stream);
  for (uint32_t i = 0; i < 50; i++) {
    ComputeStuff::Scan::compact(out_d, sum_d, scan_scratch_d, in_d, N, stream);
  }
  hipEventRecord(stopA, stream);
  hipEventSynchronize(stopA);
  hipMemcpy(out_h.data(), out_d, sizeof(uint32_t)*N, hipMemcpyDeviceToHost);

  assertMatching(sum_h, sum);
  assertMatching(out_h.data(), out.data(), sum);

  for (uint32_t i = 0; i < 10; i++) {
    ComputeStuff::HP5::compact(out_d, sum_d, hp5_scratch_d, in_d, N, stream);
  }
  hipEventRecord(startB, stream);
  for (uint32_t i = 0; i < 50; i++) {
    ComputeStuff::HP5::compact(out_d, sum_d, hp5_scratch_d, in_d, N, stream);
  }
  hipEventRecord(stopB, stream);
  hipEventSynchronize(stopB);

  //assertMatching(sum_h, sum);
  //assertMatching(out_h.data(), out.data(), sum);

  // Worst case
  buildCompactProblemWorstCase(out, sum, in, N, m);
  assertSuccess(hipMemcpy(in_d, in.data(), sizeof(uint32_t)*N, hipMemcpyHostToDevice));
  *sum_h = ~0u;
  for (uint32_t i = 0; i < 10; i++) { // Warm-up
    ComputeStuff::Scan::compact(out_d, sum_d, scan_scratch_d, in_d, N, stream);
  }
  hipEventRecord(startC, stream);
  for (uint32_t i = 0; i < 50; i++) { // Perf run
    ComputeStuff::Scan::compact(out_d, sum_d, scan_scratch_d, in_d, N, stream);
  }
  hipEventRecord(stopC, stream);
  hipEventSynchronize(stopC);
  hipMemcpy(out_h.data(), out_d, sizeof(uint32_t)*N, hipMemcpyDeviceToHost);

  assertMatching(sum_h, sum);
  assertMatching(out_h.data(), out.data(), sum);

  *sum_h = ~0u;
  for (uint32_t i = 0; i < 10; i++) { // Warm-up
    ComputeStuff::HP5::compact(out_d, sum_d, hp5_scratch_d, in_d, N, stream);
  }
  hipEventRecord(startD, stream);
  for (uint32_t i = 0; i < 50; i++) { // Perf run
    ComputeStuff::HP5::compact(out_d, sum_d, hp5_scratch_d, in_d, N, stream);
  }
  hipEventRecord(stopD, stream);
  hipEventSynchronize(stopD);

  //assertMatching(sum_h, sum);
  //assertMatching(out_h.data(), out.data(), sum);


  float elapsedA, elapsedB, elapsedC, elapsedD;
  assertSuccess(hipEventElapsedTime(&elapsedA, startA, stopA));
  assertSuccess(hipEventElapsedTime(&elapsedB, startB, stopB));
  assertSuccess(hipEventElapsedTime(&elapsedC, startC, stopC));
  assertSuccess(hipEventElapsedTime(&elapsedD, startD, stopD));

  std::cerr << std::setprecision(3)
    << "| " << N << " | "
    << (int)(100/m) << "% | "
    << (elapsedA / 50.0) << "ms | "
    << (elapsedB / 50.0) << "ms | "
    << (elapsedB / elapsedA) << " | "
    << (elapsedC / 50.0) << "ms | "
    << (elapsedD / 50.0) << "ms | "
    << (elapsedD / elapsedC) << " | " << std::endl;

  assertSuccess(hipEventDestroy(startA));
  assertSuccess(hipEventDestroy(startB));
  assertSuccess(hipEventDestroy(startC));
  assertSuccess(hipEventDestroy(startD));
  assertSuccess(hipEventDestroy(stopA));
  assertSuccess(hipEventDestroy(stopB));
  assertSuccess(hipEventDestroy(stopC));
  assertSuccess(hipEventDestroy(stopD));
  assertSuccess(hipHostFree(sum_h));
  assertSuccess(hipFree(out_d));
  assertSuccess(hipFree(in_d));
  assertSuccess(hipFree(scan_scratch_d));
  assertSuccess(hipStreamDestroy(stream));
}

int main()
{
  assertSuccess(hipSetDevice(0));

  hipDeviceProp_t props;
  assertSuccess(hipGetDeviceProperties(&props, 0));
  if (props.major < 3) {
    std::cerr << "Compute capability 3.0 is minimum." << std::endl;
    return -1;
  }


  for (uint64_t N = 1; N < (uint64_t)(props.totalGlobalMem / (sizeof(uint32_t) * 4)); N = 3 * N + N / 3) {
    for (uint32_t m = 1; m < 10; m++) {
      runCompactTest(static_cast<uint32_t>(N), m);
    }
  }

}